//dgemm testing cublas
//Stefan H
//5.12.12
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <matrix.hpp>
#include <fillMatrix.hpp>
#include <Timer.hpp>
#include <Measurement.hpp>

	

int main() {

	typedef hpc12::matrix<double,hpc12::column_major> matrix_type;
	for (int N = 512;N < 10000;N*=2) {
		matrix_type A(N,N);
		matrix_type B(N,N);
		matrix_type C(N,N);
		
		double * d_A, * d_B, *d_C;
		hipMalloc((void**) &d_A, N*N*sizeof(double));
                hipMalloc((void**) &d_B, N*N*sizeof(double));
                hipMalloc((void**) &d_C, N*N*sizeof(double));

		fillMatrix(A);
		fillMatrix(B);

		//see whether it is faster to use pinned (page locked) memory for matrices on host
		double * h_A, *h_B, *h_C;
		hipHostMalloc((void**) &h_A,N*N*sizeof(double));
		hipHostMalloc((void**) &h_B,N*N*sizeof(double));
		hipHostMalloc((void**) &h_C,N*N*sizeof(double));


		//transfer data into pinned memory
		hipMemcpy(h_A, A.data(), N*N*sizeof(double),hipMemcpyHostToHost);
		hipMemcpy(h_B, B.data(), N*N*sizeof(double),hipMemcpyHostToHost);

		// include time to copy to /from device
		Timer _t(1);

		hipMemcpy (d_A, h_A,N*N*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy (d_B, h_B ,N*N*sizeof(double),hipMemcpyHostToDevice);
		hipMemset (d_C, 0., N*N*sizeof(double));
		char trans = 'N';
		double alpha = 1.;
		double beta = 0.;
		hipblasDgemm(trans,trans,N,N,N,alpha,d_A,N,d_B,N,beta,d_C,N);
		hipMemcpy(h_C,d_C,N*N*sizeof(double),hipMemcpyDeviceToHost);
		
		_t.stop();
		Measurement m("cublasDgemm_MallocHost",N,N,_t.elapsed_s());
		std::cout << m;
		//free memory on device
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_B);

		//free pinned memory
		hipHostFree(h_A);hipHostFree(h_B);hipHostFree(h_C);
	}


	return 0;
}
