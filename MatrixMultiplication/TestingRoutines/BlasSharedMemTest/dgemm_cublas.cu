//dgemm testing cublas
//Stefan H
//5.12.12
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <matrix.hpp>
#include <fillMatrix.hpp>
#include <Timer.hpp>
#include <chrono>

	
void
dgemm_cublas(MatrixType &A, MatrixType &B, MatrixType & C) {
		//assert A columnmajor
        char transA = 'N';
        int M = A.num_rows();
        int N = B.num_cols();
        int K = A.num_cols();
        double alpha = 1.;
        int LDA = A.leading_dimension();
        int LDB = B.leading_dimension();
        double beta = 0.;
        int LDC = C.leading_dimension();
        hipblasDgemm(transA,transA,M,N,K,alpha,A.data(),LDA,B.data(),LDB,beta,C.data(),LDC);

}


int main() {

	typedef hpc12::matrix<double,hpc12::column_major> matrix_type;
	for (int N = 512;N < 20000;N*=2) {
		matrix_type A(N,N);
		matrix_type B(N,N);
		matrix_type C(N,N);
		fillMatrix(A);
		fillMatrix(B);
		//C = A*B
		Timer _t(1);
		dgemm_libsci(A,B,C);
		_t.stop();
		Measurement m("dgemm with libsci,?noacc?,16thrds",N,N,_t.elapsed_s());
		std::cout << m;
	}


	return 0;
}
