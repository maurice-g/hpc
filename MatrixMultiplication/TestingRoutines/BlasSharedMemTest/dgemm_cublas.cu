//dgemm testing cublas
//Stefan H
//5.12.12
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <matrix.hpp>
#include <fillMatrix.hpp>
#include <Timer.hpp>
#include <Measurement.hpp>

	

int main() {

	typedef hpc12::matrix<double,hpc12::column_major> matrix_type;
	for (int N = 512;N < 10000;N*=2) {
		matrix_type A(N,N);
		matrix_type B(N,N);
		matrix_type C(N,N);
		
		double * d_A, * d_B, *d_C;
		hipMalloc((void**) &d_A, N*N*sizeof(double));
                hipMalloc((void**) &d_B, N*N*sizeof(double));
                hipMalloc((void**) &d_C, N*N*sizeof(double));

		fillMatrix(A);
		fillMatrix(B);

		Timer _t(1);

		hipMemcpy (d_A, A.data(),N*N*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy (d_B, B.data(),N*N*sizeof(double),hipMemcpyHostToDevice);
		hipMemset (d_C, 0., N*N*sizeof(double));
		//C = A*B
		char trans = 'N';
		double alpha = 1.;
		double beta = 0.;
		hipblasDgemm(trans,trans,N,N,N,alpha,d_A,N,d_B,N,beta,d_C,N);
//dgemm_cublas(d_A,d_B,d_C);
		hipMemcpy(C.data(),d_C,N*N*sizeof(double),hipMemcpyDeviceToHost);
		
		_t.stop();
		Measurement m("hipblasDgemm",N,N,_t.elapsed_s());
		std::cout << m;
		//free memory on device
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_B);
	}


	return 0;
}
