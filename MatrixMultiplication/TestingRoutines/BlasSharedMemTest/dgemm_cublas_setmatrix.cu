//dgemm testing cublas version 2, using set /get matrix for transferring data d2h h2d
//Stefan H
//13.12.12
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <matrix.hpp>
#include <fillMatrix.hpp>
#include <Timer.hpp>
#include <Measurement.hpp>

	

int main() {
	hipblasHandle_t handle;
	hipblasStatus_t status = hipblasCreate(&handle);

	typedef hpc12::matrix<double,hpc12::column_major> matrix_type;
	for (int N = 512;N < 10000;N*=2) {
		matrix_type A(N,N);
		matrix_type B(N,N);
		matrix_type C(N,N);
		
		fillMatrix(A);
		fillMatrix(B);

		
		double * d_A, * d_B, *d_C;
		hipMalloc((void**) &d_A, N*N*sizeof(double));
        hipMalloc((void**) &d_B, N*N*sizeof(double));
        hipMalloc((void**) &d_C, N*N*sizeof(double));

		//allocate pinned memory
		double * h_A, *h_B, *h_C;
		hipHostMalloc((void**) &h_A,N*N*sizeof(double));
		hipHostMalloc((void**) &h_B,N*N*sizeof(double));
		hipHostMalloc((void**) &h_C,N*N*sizeof(double));



		//transfer data into pinned memory
		hipMemcpy(h_A, A.data(), N*N*sizeof(double),hipMemcpyHostToHost);
		hipMemcpy(h_B, B.data(), N*N*sizeof(double),hipMemcpyHostToHost);

		
		Timer _t(1);

		hipblasSetMatrix (A.num_rows(),A.num_cols(),sizeof(double),h_A,A.leading_dimension(),d_A,A.leading_dimension());
		hipblasSetMatrix (B.num_rows(),B.num_cols(),sizeof(double),h_B,B.leading_dimension(),d_B,B.leading_dimension());
		hipMemset (d_C, 0., N*N*sizeof(double));
		//C = A*B
		double alpha = 1.;
		double beta = 0.;
		
		hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,N,N,N,&alpha,d_A,N,d_B,N,&beta,d_C,N);
		
		hipblasGetMatrix(C.num_rows(),C.num_cols(),sizeof(double),d_C,C.leading_dimension(),h_C,C.leading_dimension());
		
		
		_t.stop();
		Measurement m("hipblasDgemm v2, setmatrix etc",N,N,_t.elapsed_s());
		std::cout << m;
		//free memory on device
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		
		hipHostFree(h_A);hipHostFree(h_B);hipHostFree(h_C);
	}
	hipblasDestroy(handle);

	return 0;
}
