//dgemm testing cublas, measuring flops with and without copy to dev and membdwdth
//Stefan H
//11.12.12
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <matrix.hpp>
#include <fillMatrix.hpp>
#include <Timer.hpp>
#include <Measurement.hpp>

#ifndef FLOPSWITHOUTHCPY
#define FLOPSWITHOUTCPY 0
#endif

#ifndef MEMBDWDTH
#define MEMBDWDTH 0
#endif

#ifndef FLOPS
#define FLOPS 0
#endif
	

int main() {
	std::cout << "performance  of cublas dgemm using pinned memory \n";
	typedef hpc12::matrix<double,hpc12::column_major> matrix_type;
	for (int N = 512;N < 10000;N*=2) {
		matrix_type A(N,N);
		matrix_type B(N,N);
		matrix_type C(N,N);
		
		double * d_A, * d_B, *d_C;
		hipMalloc((void**) &d_A, N*N*sizeof(double));
                hipMalloc((void**) &d_B, N*N*sizeof(double));
                hipMalloc((void**) &d_C, N*N*sizeof(double));

		fillMatrix(A);
		fillMatrix(B);

		//see whether it is faster to use pinned (page locked) memory for matrices on host
		double * h_A, *h_B, *h_C;
		hipHostMalloc((void**) &h_A,N*N*sizeof(double));
		hipHostMalloc((void**) &h_B,N*N*sizeof(double));
		hipHostMalloc((void**) &h_C,N*N*sizeof(double));


		//transfer data into pinned memory
		hipMemcpy(h_A, A.data(), N*N*sizeof(double),hipMemcpyHostToHost);
		hipMemcpy(h_B, B.data(), N*N*sizeof(double),hipMemcpyHostToHost);

		// include time to copy to /from device
		
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		//hipEventRecord(start,0);

		hipMemcpy (d_A, h_A,N*N*sizeof(double),hipMemcpyHostToDevice);
		hipMemcpy (d_B, h_B ,N*N*sizeof(double),hipMemcpyHostToDevice);

		//hipEventRecord(stop,0);
		//hipEventSynchronize(stop);
		float elapsed_time;
		double elapsed;
		/*hipEventElapsedTime(&elapsed_time,start,stop);//time in miliseconds, bw in bytes: -->GB/s: 1e-9*1e3
		std::cout << N  << "Peak memory bandwidth H2D(GB/s): " << 1e-6*2*N*N*sizeof(double)/elapsed_time << "\n";
		hipEventDestroy(start);
		hipEventDestroy(stop);
*/
		hipMemset (d_C, 0., N*N*sizeof(double));
		char trans = 'N';
		double alpha = 1.;
		double beta = 0.;
		hipEventRecord(start,0);
		hipblasDgemm(trans,trans,N,N,N,alpha,d_A,N,d_B,N,beta,d_C,N);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time,start,stop);elapsed=elapsed_time;
		
		//craaazy bug: try double(N*N*N)/(elapsed*0.001) ---> result is 0 WTF??
		std::cout << N << " runtime: "<< elapsed  << " peak FLOPS without copying data: " << double(N)*double(N)*double(N)/(elapsed*0.001) <<"\n";
		

		hipMemcpy(h_C,d_C,N*N*sizeof(double),hipMemcpyDeviceToHost);		

		//free memory on device
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_B);

		//free pinned memory
		hipHostFree(h_A);hipHostFree(h_B);hipHostFree(h_C);
	}


	return 0;
}
